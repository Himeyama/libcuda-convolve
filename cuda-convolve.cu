#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <algorithm>
#include <iterator>
#define max(a, b) ((a > b) ? a : b)

template <typename T>
__global__
void cuda_convolve_full(T *a, T *v, T *conv, long a_size, long v_size){
    long i = blockDim.x * blockIdx.x + threadIdx.x;
    conv[i] = 0;
    if(i < a_size + v_size - 1)
        for(long j = 0; j < v_size; j++)
            conv[i] += v[j] * ((i - j >= a_size || i - j < 0) ? 0 : a[i - j]);
}

template <typename T>
T* fary2cuda(std::vector<T> a){
    T *g;
    hipError_t err = hipMalloc((void**)&g, sizeof(T) * a.size());
    if(err) exit(err);
    hipMemcpy(g, a.data(), sizeof(T) * a.size(), hipMemcpyHostToDevice);
    return g;
}

template <typename T>
std::vector<T> convolve(std::vector<T> a, std::vector<T> v){
    std::vector<T> conv(a.size() + v.size() - 1);
    T* ga = fary2cuda(a);
    T* gv = fary2cuda(v);
    T* gconv = fary2cuda(conv);
    cuda_convolve_full<<<(a.size()+256)/256, 256>>>(ga, gv, gconv, a.size(), v.size());
    hipDeviceSynchronize();
    hipMemcpy(conv.data(), gconv, sizeof(T) * conv.size(), hipMemcpyDeviceToHost);
    hipFree(ga);
    hipFree(gv);
    hipFree(gconv);
    hipDeviceReset();
    return conv;
}

template <typename T>
std::vector<T> convolve_same(std::vector<T> a, std::vector<T> v){
    std::vector<T> conv(max(a.size(), v.size()));
    long sidx = (int)round(v.size() / 2.0) - 1;
    std::vector<T> conv_full = convolve(a, v);
    memcpy(conv.data(), conv_full.data() + sidx, sizeof(T) * conv.size());
    return conv;
}

template <typename T>
std::vector<T> convolve_valid(std::vector<T> a, std::vector<T> v){
    std::vector<T> conv(max(a.size(), v.size()) - min(a.size(), v.size()) + 1);
    long sidx = v.size() - 1;
    std::vector<T> conv_full = convolve(a, v);
    memcpy(conv.data(), conv_full.data() + sidx, sizeof(T) * conv.size());
    return conv;
}

template std::vector<float> convolve<float>(std::vector<float>, std::vector<float>);
template std::vector<float> convolve_valid<float>(std::vector<float>, std::vector<float>);
template std::vector<float> convolve_same<float>(std::vector<float>, std::vector<float>);

template std::vector<double> convolve<double>(std::vector<double>, std::vector<double>);
template std::vector<double> convolve_valid<double>(std::vector<double>, std::vector<double>);
template std::vector<double> convolve_same<double>(std::vector<double>, std::vector<double>);
