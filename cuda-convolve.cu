#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <algorithm>
#include <iterator>
#define max(a, b) ((a > b) ? a : b)

__global__
void cuda_convolve_full(float *a, float *v, float *conv, long a_size, long v_size){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    conv[i] = 0;
    if(i < a_size + v_size - 1)
        for(long j = 0; j < v_size; j++)
            conv[i] += v[j] * ((i - j >= a_size || i - j < 0) ? 0 : a[i - j]);
}

template <typename T>
T* fary2cuda(std::vector<T> a){
    T *g;

    hipError_t err = hipMalloc((void**)&g, sizeof(T) * a.size());
    if(err){
        std::cout << err << std::endl;
        std::cout << hipGetErrorString(err) << std::endl;
    }
    hipMemcpy(g, a.data(), sizeof(T) * a.size(), hipMemcpyHostToDevice);
    return g;
}

template <typename T>
std::vector<T> convolve(std::vector<T> a, std::vector<T> v){
    std::vector<float> conv(a.size() + v.size() - 1);
    T* ga = fary2cuda(a);
    T* gv = fary2cuda(v);
    T* gconv = fary2cuda(conv);
    cuda_convolve_full<<<(a.size()+256)/256, 256>>>(ga, gv, gconv, a.size(), v.size());
    hipDeviceSynchronize();
    hipMemcpy(conv.data(), gconv, sizeof(T) * conv.size(), hipMemcpyDeviceToHost);
    hipFree(ga);
    hipFree(gv);
    hipFree(gconv);
    hipDeviceReset();
    return conv;
}

template <typename T>
std::vector<T> convolve_same(std::vector<T> a, std::vector<T> v){
    std::vector<T> conv(max(a.size(), v.size()));
    long sidx = (int)round(v.size() / 2.0) - 1;
    std::vector<T> conv_full = convolve(a, v);
    memcpy(conv.data(), conv_full.data() + sidx, sizeof(T) * conv.size());
    return conv;
}

template <typename T>
std::vector<T> convolve_valid(std::vector<T> a, std::vector<T> v){
    std::vector<T> conv(max(a.size(), v.size()) - min(a.size(), v.size()) + 1);
    long sidx = v.size() - 1;
    std::vector<T> conv_full = convolve(a, v);
    memcpy(conv.data(), conv_full.data() + sidx, sizeof(T) * conv.size());
    return conv;
}

template std::vector<float> convolve<float>(std::vector<float>, std::vector<float>);
template std::vector<float> convolve_valid<float>(std::vector<float>, std::vector<float>);
template std::vector<float> convolve_same<float>(std::vector<float>, std::vector<float>);