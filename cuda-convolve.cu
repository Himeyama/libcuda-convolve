#include "hip/hip_runtime.h"
#include "float-vector.h"
#define max(a, b) ((a > b) ? a : b)

__global__
void cuda_convolve_full(float *a, float *v, float *conv, FloatVec vec_a, FloatVec vec_v){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    conv[i] = 0;
    if(i < vec_a.size + vec_v.size - 1)
        for(int j = 0; j < vec_v.size; j++)
            conv[i] += v[j] * ((i - j >= vec_a.size || i - j < 0) ? 0 : a[i - j]);
}

FloatVec convolve(FloatVec a, FloatVec v){
    FloatVec conv;
    init_FloatVec(&conv, a.size + v.size - 1);
    float* ga = fary2cuda(a);
    float* gv = fary2cuda(v);
    float* gconv = fary2cuda(conv);
    cuda_convolve_full<<<(a.size+256)/256, 256>>>(ga, gv, gconv, a, v);
    hipDeviceSynchronize();
    hipMemcpy(conv.data, gconv, conv.bytes, hipMemcpyDeviceToHost);
    hipFree(ga);
    hipFree(gv);
    hipFree(gconv);
    return conv;
}

FloatVec convolve_same(FloatVec a, FloatVec v){
    FloatVec conv;
    long sidx = (int)round(v.size / 2.0) - 1;
    init_FloatVec(&conv, max(a.size, v.size));
    FloatVec conv_full = convolve(a, v);
    memcpy(conv.data, conv_full.data + sidx, conv.bytes);
    free(conv_full.data);
    return conv;
}

FloatVec convolve_valid(FloatVec a, FloatVec v){
    FloatVec conv;
    long sidx = v.size - 1;
    init_FloatVec(&conv, max(a.size, v.size) - min(a.size, v.size) + 1);
    FloatVec conv_full = convolve(a, v);
    memcpy(conv.data, conv_full.data + sidx, conv.bytes);
    free(conv_full.data);
    return conv;
}

extern "C" {
    void test(){
        FloatVec a, v;
        init_FloatVec(&a, 3);
        for(int i = 0; i < a.size; i++)
            a.data[i] = i + 1; 
        init_FloatVec(&v, 3);
        v.data[0] = 0;
        v.data[1] = 1;
        v.data[2] = 0.5;
        FloatVec conv = convolve(a, v);
        printVec(conv);
        free(conv.data);

        conv = convolve_same(a, v);
        printVec(conv);
        free(conv.data);

        conv = convolve_valid(a, v);
        printVec(conv);

        free(conv.data);
        free(v.data);
        free(a.data);
    }
}