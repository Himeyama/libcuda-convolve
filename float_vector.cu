#include <stdio.h>
#include "float_vector.h"

void init_FloatVec(FloatVec *a, long size){
    a->size = size;
    a->data = (float*)malloc(sizeof(float) * size);
    a->bytes = sizeof(float) * size;
    memset(a->data, 0, a->bytes);
}

void printVec(FloatVec a){
    char *tmp = (char*)malloc(a.size * 24 + 3);
    tmp[0] = '[';
    tmp[1] = '\0';
    char number[12];
    for(int i = 0; i < a.size; i++){
        if(i < a.size - 1)
            sprintf(number, "%f, ", a.data[i]);
        else
            sprintf(number, "%f]", a.data[i]);
        strcat(tmp, number);
    }
    puts(tmp);
    free(tmp);
}

float* fary2cuda(FloatVec a){
    float *g;
    hipMalloc((void**)&g, a.bytes);
    hipMemcpy(g, a.data, a.bytes, hipMemcpyHostToDevice);
    return g;
}